#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <memory>

#include <hip/hip_runtime.h>
#include <>

#define __COMPILE_CUDA__
#include <oc/arrnd.h>

#define WARMUP_KERNEL_RESULT (.5f)
__global__ void warmup_kernel(float* p)
{
    unsigned int idx = threadIdx.x;
    p[idx] = .5f;
}

class cuda_tests : public ::testing::Test {
public:
    static void SetUpTestCase()
    {
        hipError_t err;

        err = hipSetDevice(device_id);
        if (err != hipSuccess) {
            FAIL() << "Failed to set CUDA device " << device_id << "(" << err << ": " << hipGetErrorString(err) << ")";
        }

        float h_ref = -1.f;
        err = warmup(h_ref);
        if (err != hipSuccess) {
            FAIL() << "Failed to wramup CUDA device (" << err << ": " << hipGetErrorString(err) << ")";
        }
        if (h_ref != WARMUP_KERNEL_RESULT) {
            FAIL() << "Invalid warmup kernel result (required: " << WARMUP_KERNEL_RESULT << ", actual: " << h_ref
                   << ")";
        }
    }

    static void TearDownTestCase()
    {
        hipError_t err = hipDeviceReset();
        if (err != hipSuccess) {
            FAIL() << "Failed to reset CUDA device (" << err << ": " << hipGetErrorString(err) << ")";
        }
    }

private:
    // simple CUDA GPU warmup 
    static hipError_t warmup(float& h_ref)
    {
        std::unique_ptr<float> h_p = std::make_unique<float>(0.f);

        hipError_t err;

        float* d_p;
        err = hipMalloc((void**)&d_p, sizeof(float));
        if (err != hipSuccess) {
            return err;
        }
        err = hipMemcpy(d_p, h_p.get(), sizeof(float), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            return err;
        }

        warmup_kernel<<<1, 1>>>(d_p);
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            return err;
        }

        err = hipMemcpy(h_p.get(), d_p, sizeof(float), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            return err;
        }
        h_ref = *h_p;

        hipFree(d_p);

        return hipSuccess;
    }

    static constexpr int device_id = 0;
};

TEST_F(cuda_tests, dummy)
{
    SUCCEED();
}
