#include <gtest/gtest.h>

#include <hip/hip_runtime.h>

#define __COMPILE_CUDA__
#include <oc/arrnd.h>

TEST(cuda, dummy_kernel)
{
    hipError_t err;
    dummy_kernel<<<1, 1>>>();
    err = hipGetLastError();
    EXPECT_EQ(err, hipSuccess);
    err = hipDeviceSynchronize();
    EXPECT_EQ(err, hipSuccess);
}
